#include "GlobalVariables.hpp"
#include "BFSInit.cuh"
#include "BFSGPU.cuh"

void BFSsequential(const vector<int>& startEdge, const vector<int>& endEdge, int u, int v, ostream& file) {
	int numVertices = *max_element(endEdge.begin(), endEdge.end()) + 1;
	vector<vector<int>> adjacencyList(numVertices);
	for (size_t i = 0; i < startEdge.size(); ++i) {
		adjacencyList[startEdge[i]].push_back(endEdge[i]);
	}

	vector<bool> visited(numVertices, false);
	queue<int> que;
	que.push(u);
	visited[u] = true;
	int* h_prev = new int[numVertices];
	for (int i = 0; i < numVertices; ++i)
		h_prev[i] = -1;
	while (!que.empty() && !visited[v]) {
		int currentVertex = que.front();
		que.pop();
		// Przeszukiwanie s�siad�w
		for (int neighbor : adjacencyList[currentVertex]) {
			if (!visited[neighbor]) {
				h_prev[neighbor] = currentVertex;
				visited[neighbor] = true;
				que.push(neighbor);
			}
		}
	}
	vector<int> path;
	findPath(h_prev, u, v, path);
	if (!path.empty())
	{
		file << "\n Sequential BFS\n";
		std::copy(path.begin(), path.end(), std::ostream_iterator<int>(file, " "));
		file << "\n";
	}
	else
	{
		file << "Nie znaleziono sciezki. -- Algorytm sekwencyjny\n";
	}
	delete[] h_prev;
}
void findPath(int* prev, int u, int v, std::vector<int>& path) {
	int current = v;
	path.clear();

	while (current != -1 && current != u) {
		path.push_back(current);
		current = prev[current];
	}

	if (current == u) {
		path.push_back(u);
		std::reverse(path.begin(), path.end());
	}
	else {
		path.clear();
	}
}

// Funkcja uruchamiaj�ca algorytmy BFS na GPU
hipError_t BFSCuda(vector<int>& startEdge, vector<int>& endEdge, int u, int v, ofstream& file) {
	// inicjalizuje zmienne
	hipError_t cudaStatus;
	hipEvent_t startEvent, stopEvent;
	float elapsedMilliseconds;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);
	hipEventRecord(startEvent, 0);
	int size = *thrust::max_element(endEdge.begin(), endEdge.end());
	int iteration = 0;
	int* frontier = new int[size + 1];
	frontier[u] = 1;
	thrust::device_vector<int> d_size_of_new_queue(1);
	thrust::device_vector<int> d_size_of_old_queue(1);
	thrust::device_vector<int> d_startEdge(startEdge.begin(), startEdge.end());
	thrust::device_vector<int> d_endEdge(endEdge.begin(), endEdge.end());
	thrust::device_vector<int> d_queue(size);
	thrust::device_vector<int> d_prefix_sums(size + 1);
	thrust::device_vector<int> d_frontier(size + 1);
	thrust::device_vector<int> output(size);
	thrust::device_vector<int> begin_list(d_startEdge.size());
	thrust::device_vector<int> adj_vector(d_startEdge.size());
	thrust::device_vector<int> distance(size + 1);
	thrust::device_vector<int> d_newFrontier(size + 1);
	thrust::device_vector<int> d_prev(size + 1);
	bool change = 1, * d_change, * d_possible, possible = true;
	int block_size = BLOCK_SIZE;
	int num_blocks = (size + block_size - 1) / block_size;
	int block_size_edges = BLOCK_SIZE;
	int num_blocks_edges = (d_startEdge.size() + block_size_edges - 1) / block_size_edges;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_change, sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for d_change: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&d_possible, sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for d_possible: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	hipMemcpy(d_change, &change, sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for d_change!");
		goto Error;
	}

	hipMemcpy(d_possible, &possible, sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for d_possible!");
		goto Error;
	}

	// ustawiam poczatkowa dlugosc kolejki
	thrust::fill(d_size_of_old_queue.begin(), d_size_of_old_queue.end(), 1);
	// ustawiam poprzednikow wierzcholkow
	thrust::fill(d_prev.begin(), d_prev.end(), -1);
	// tworze poczatkowa kolejke
	thrust::fill(d_queue.begin(), d_queue.begin() + 1, u);
	thrust::fill(d_queue.begin() + 1, d_queue.end(), -1);
	// ustawiam odleglosci do wierzcholkow
	thrust::fill(distance.begin(), distance.begin() + u, MAX_DISTANCE);
	thrust::fill(distance.begin() + u + 1, distance.end(), MAX_DISTANCE);
	// ustawiam tablcie wierzcholkow ktore maja byc odwiedzone w kolejnym kroku
	thrust::fill(d_frontier.begin(), d_frontier.begin() + u, 0);
	thrust::fill(d_frontier.begin() + u + 1, d_frontier.end(), 0);
	thrust::fill(d_frontier.begin() + u, d_frontier.begin() + u + 1, 1);

	// tworze tablice gdzie mam posortowane krawedzie (poczatki i konce w drugiej tablicy), a nastepnie obliczam gdzie zaczyna sie kazdy wierzcholek
	thrust::sort_by_key(d_startEdge.begin(), d_startEdge.end(), d_endEdge.begin());
	thrust::adjacent_difference(d_startEdge.begin(), d_startEdge.end(), adj_vector.begin());
	kernel_cuda_generate_begin_list << <num_blocks_edges, block_size_edges >> > (thrust::raw_pointer_cast(d_startEdge.data()), thrust::raw_pointer_cast(adj_vector.data()), thrust::raw_pointer_cast(begin_list.data()), d_startEdge.size());
	if (hipGetLastError() != hipSuccess)
		goto Error;
	if (hipDeviceSynchronize() != hipSuccess)
		goto Error;
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedMilliseconds, startEvent, stopEvent);
	std::cout << "Initialize Time: " << elapsedMilliseconds << " ms\n" << std::endl;

	// Wersja z kolejka bez atomowych operacji
	hipEventRecord(startEvent, 0);
	cudaStatus = BFSQueueVersion(d_startEdge, d_endEdge, d_queue, d_prefix_sums, d_frontier, d_newFrontier, d_change, size, block_size, num_blocks, iteration, begin_list, distance, d_startEdge.size(), v, d_prev, file, u);
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedMilliseconds, startEvent, stopEvent);
	std::cout << "BFSQueueVersion Time: " << elapsedMilliseconds << " ms\n" << std::endl;

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "BFSQueueVersion failed! Error: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Wersja bez kolejki, za kazdym razem przegladam nieodwiedzonych sasiadow
	hipEventRecord(startEvent, 0);
	cudaStatus = BFSLayersVersion(d_startEdge, d_endEdge, d_frontier, d_newFrontier, d_change, size, block_size, num_blocks, iteration, begin_list, distance, d_startEdge.size(), v, d_prev, file, u);
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedMilliseconds, startEvent, stopEvent);
	std::cout << "BFSLayersVersion Time: " << elapsedMilliseconds << " ms\n" << std::endl;

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "BFSLayersVersion failed! Error: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Wersja z kolejkami lokalnymi z atomowymi operacjami (nie dziala gdy graf ma duzo krawedzi, nie miesci sie w shared memory)
	hipEventRecord(startEvent, 0);
	cudaStatus = BFSAtomicOppVersion(d_startEdge, d_endEdge, d_queue, d_size_of_new_queue, d_size_of_old_queue, d_change, size, block_size, num_blocks, begin_list, distance, d_startEdge.size(), iteration, v, d_prev, file, u, d_possible);
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedMilliseconds, startEvent, stopEvent);
	std::cout << "BFSAtomicOppVersion Time: " << elapsedMilliseconds << " ms\n" << std::endl;

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "BFSAtomicOppVersion failed! Error: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Wersja z kolejka globalna z atomowymi operacjami
	hipEventRecord(startEvent, 0);
	cudaStatus = BFSAtomicOppGlobalVersion(d_startEdge, d_endEdge, d_queue, d_size_of_old_queue, d_change, size, block_size, num_blocks, begin_list, distance, d_startEdge.size(), iteration, v, d_prev, file, u);
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedMilliseconds, startEvent, stopEvent);
	std::cout << "BFSAtomicOppGlobalVersion Time: " << elapsedMilliseconds << " ms\n" << std::endl;

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "BFSAtomicOppGlobalVersion failed! Error: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	delete[] frontier;
	hipFree(d_change);
	hipFree(d_possible);
	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);

	return cudaStatus;
}
hipError_t BFSQueueVersion(thrust::device_vector<int> d_startEdge, thrust::device_vector<int> d_endEdge, thrust::device_vector<int> d_queue, thrust::device_vector<int> d_prefix_sums, thrust::device_vector<int> d_frontier, thrust::device_vector<int> d_newFrontier, bool* d_change, int size, int block_size, int num_blocks, int iteration, thrust::device_vector<int> begin_list, thrust::device_vector<int> distance, int number_of_edges, int v, thrust::device_vector<int> d_prev, ofstream& file, int u)
{
	hipError_t cudaStatus = hipSuccess;
	bool change = true;
	int distance_v = MAX_DISTANCE;
	hipEvent_t startQueueEvent, stopQueueEvent, startLoopEvent, stopLoopEvent;
	float elapsedQueueMilliseconds_Sum = 0.0f;
	float elapsedLoopMilliseconds_Sum = 0.0f;
	int* d_queue_act_size;
	int queue_act_size = 1;
	cudaStatus = hipMalloc((void**)&d_queue_act_size, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for d_change: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}
	hipEventCreate(&startQueueEvent);
	hipEventCreate(&stopQueueEvent);
	hipEventCreate(&startLoopEvent);
	hipEventCreate(&stopLoopEvent);
	// petla ktora dziala do momentu az odleglosc do ostatniego wierzcholka sie zmieni
	while (distance_v == MAX_DISTANCE)
	{
		// poczatek mierzenia czasu tworzenia kolejki
		hipEventRecord(startQueueEvent, 0);
		// tworze tablice sum prefiksowych, aby pozniej wiedziec ile jest elementow w kolejce i latwo obliczyc miejsce w kolejce
		thrust::exclusive_scan(d_frontier.begin(), d_frontier.begin() + size, d_prefix_sums.begin());
		// tworze kolejke
		kernel_cuda_generate_queue << <num_blocks, block_size >> > (thrust::raw_pointer_cast(d_prefix_sums.data()), thrust::raw_pointer_cast(d_frontier.data()), thrust::raw_pointer_cast(d_queue.data()), size);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
			return cudaStatus;

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)
			return cudaStatus;
		hipEventRecord(stopQueueEvent, 0);
		hipEventSynchronize(stopQueueEvent);
		float elapsedQueueMilliseconds;
		hipEventElapsedTime(&elapsedQueueMilliseconds, startQueueEvent, stopQueueEvent);
		elapsedQueueMilliseconds_Sum += elapsedQueueMilliseconds;
		hipMemcpy(&queue_act_size, thrust::raw_pointer_cast(&(d_queue[0])), sizeof(int), hipMemcpyDeviceToHost);
		// jezeli kolejka jest pusta koncze wykonywanie petli
		if (queue_act_size == 0)
		{
			break;
		}
		// poczatek mierzenia czasu przegladania sasiadow
		hipEventRecord(startLoopEvent, 0);
		int num_b = (queue_act_size + block_size - 1) / block_size;
		// przegladam sasiadow wierzcholkow znajdujacych sie w kolejce
		BFSPrescan << <num_b, block_size >> > (iteration++, thrust::raw_pointer_cast(d_queue.data()), size, thrust::raw_pointer_cast(d_startEdge.data()), thrust::raw_pointer_cast(d_endEdge.data()), thrust::raw_pointer_cast(begin_list.data()), thrust::raw_pointer_cast(distance.data()), thrust::raw_pointer_cast(d_newFrontier.data()), d_change, d_startEdge.size(), thrust::raw_pointer_cast(d_prev.data()));

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
			return cudaStatus;

		// przygotowuje tablice Frontier do stworzenia kolejki w kolejnym kroku
		thrust::copy(d_newFrontier.begin(), d_newFrontier.begin() + size, d_frontier.begin());
		thrust::fill(d_newFrontier.begin(), d_newFrontier.begin() + size, 0);
		hipMemcpy(&change, d_change, sizeof(bool) * 1, hipMemcpyDeviceToHost);
		hipEventRecord(stopLoopEvent, 0);
		hipEventSynchronize(stopLoopEvent);
		float elapsedLoopMilliseconds;
		hipEventElapsedTime(&elapsedLoopMilliseconds, startLoopEvent, stopLoopEvent);
		elapsedLoopMilliseconds_Sum += elapsedLoopMilliseconds;
		hipMemcpy(&distance_v, thrust::raw_pointer_cast(&(distance[v])), sizeof(int), hipMemcpyDeviceToHost);
	}

	std::cout << "Calkowity czas tworzenia kolejki: " << elapsedQueueMilliseconds_Sum << " ms" << std::endl;
	std::cout << "Calkowity czas BFSPrescan'u: " << elapsedLoopMilliseconds_Sum << " ms" << std::endl;

	hipEventDestroy(startQueueEvent);
	hipEventDestroy(stopQueueEvent);
	hipEventDestroy(startLoopEvent);
	hipEventDestroy(stopLoopEvent);
	vector<int> path;

	int* h_prev = new int[size + 1];
	hipMemcpy(h_prev, thrust::raw_pointer_cast(d_prev.data()), (size + 1) * sizeof(int), hipMemcpyDeviceToHost);

	// Znajdz i wypisz sciezke
	findPath(h_prev, u, v, path);

	if (!path.empty())
	{
		file << "\nQueue BFS\n";
		std::copy(path.begin(), path.end(), std::ostream_iterator<int>(file, " "));
		file << "\n";
	}
	else
	{
		file << "Nie znaleziono sciezki. -- Algorytm CUDA BFS z kolejka\n";
	}

	delete[] h_prev;
	return cudaStatus;
}
hipError_t BFSLayersVersion(thrust::device_vector<int> d_startEdge, thrust::device_vector<int> d_endEdge, thrust::device_vector<int> d_frontier, thrust::device_vector<int> d_newFrontier, bool* d_change, int size, int block_size, int num_blocks, int iteration, thrust::device_vector<int> begin_list, thrust::device_vector<int> distance, int number_of_edges, int v, thrust::device_vector<int> d_prev, ofstream& file, int u)
{
	bool change = true;
	int distance_v = MAX_DISTANCE;
	hipError_t cudaStatus = hipSuccess;
	// petla ktora dziala do momentu az ostatni wierzcholek jest nieodwiedzony i jest jeszcze nieodwiedzony sasaid
	while (change && distance_v == MAX_DISTANCE)
	{
		change = 0;
		hipMemcpy(d_change, &change, sizeof(bool) * 1, hipMemcpyHostToDevice);
		// przegladam nieodwiedzonych sasiadow
		BFSLayers << <num_blocks, block_size >> > (iteration++, thrust::raw_pointer_cast(d_frontier.data()), size, thrust::raw_pointer_cast(d_startEdge.data()), thrust::raw_pointer_cast(d_endEdge.data()), thrust::raw_pointer_cast(begin_list.data()), thrust::raw_pointer_cast(distance.data()), thrust::raw_pointer_cast(d_newFrontier.data()), d_change, d_startEdge.size(), thrust::raw_pointer_cast(d_prev.data()));

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
			return cudaStatus;

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)
			return cudaStatus;
		// przygotowuje tablcie Frontier do nastepnego kroku
		thrust::copy(d_newFrontier.begin(), d_newFrontier.end(), d_frontier.begin());
		thrust::fill(d_newFrontier.begin(), d_newFrontier.end(), 0);
		hipMemcpy(&change, d_change, sizeof(bool) * 1, hipMemcpyDeviceToHost);
		hipMemcpy(&distance_v, thrust::raw_pointer_cast(&(distance[v])), sizeof(int), hipMemcpyDeviceToHost);
	}
	vector<int> path;
	int* h_prev = new int[size + 1];
	hipMemcpy(h_prev, thrust::raw_pointer_cast(d_prev.data()), (size + 1) * sizeof(int), hipMemcpyDeviceToHost);

	// Znajdz i wypisz sciezke
	findPath(h_prev, u, v, path);
	if (!path.empty())
	{
		file << "\nFrontiers BFS\n";
		std::copy(path.begin(), path.end(), std::ostream_iterator<int>(file, " "));
		file << "\n";
	}
	else
	{
		file << "Nie znaleziono sciezki. -- Algorytm CUDA BFS Frontiers\n";
	}

	delete[] h_prev;
	return cudaStatus;
}
hipError_t BFSAtomicOppVersion(thrust::device_vector<int> d_startEdge, thrust::device_vector<int> d_endEdge, thrust::device_vector<int> d_queue, thrust::device_vector<int> d_size_of_new_queue, thrust::device_vector<int> d_size_of_old_queue, bool* d_change, int size, int block_size, int num_blocks, thrust::device_vector<int> begin_list, thrust::device_vector<int> distance, int number_of_edges, int iteration, int v, thrust::device_vector<int> d_prev, ofstream& file, int u, bool* d_possible)
{
	hipError_t cudaStatus = hipSuccess;
	bool change = true;
	bool possible = true;
	int zero = 0, distance_v = MAX_DISTANCE;
	// petla ktora dziala do momentu az ostatni wierzcholek jest nieodwiedzony i jest jeszcze nieodwiedzony sasiad
	while (change && possible && distance_v == MAX_DISTANCE) {
		change = false;
		hipMemcpy(d_change, &change, sizeof(bool), hipMemcpyHostToDevice);

		// wlaczam BFS z atomowymi operacjami i lokalna kolejka
		BFSAtomicOpp << <num_blocks, block_size >> > (
			thrust::raw_pointer_cast(d_queue.data()),
			size,
			thrust::raw_pointer_cast(d_startEdge.data()),
			thrust::raw_pointer_cast(d_endEdge.data()),
			thrust::raw_pointer_cast(begin_list.data()),
			thrust::raw_pointer_cast(distance.data()),
			d_change,
			d_startEdge.size(),
			thrust::raw_pointer_cast(d_size_of_new_queue.data()),
			thrust::raw_pointer_cast(d_size_of_old_queue.data()),
			iteration++, thrust::raw_pointer_cast(d_prev.data()),
			d_possible
			);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
			return cudaStatus;

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)
			return cudaStatus;

		hipMemcpy(&change, d_change, sizeof(bool), hipMemcpyDeviceToHost);
		hipMemcpy(d_size_of_old_queue.data().get(), d_size_of_new_queue.data().get(), sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(d_size_of_new_queue.data().get(), &zero, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(&distance_v, thrust::raw_pointer_cast(&(distance[v])), sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&possible, d_possible, sizeof(bool), hipMemcpyDeviceToHost);
	}

	if (possible)
	{
		vector<int> path;
		int* h_prev = new int[size + 1];
		hipMemcpy(h_prev, thrust::raw_pointer_cast(d_prev.data()), (size + 1) * sizeof(int), hipMemcpyDeviceToHost);

		// Znajdz i wypisz sciezke
		findPath(h_prev, u, v, path);
		if (!path.empty())
		{
			file << "\nAtomic Opperations BFS\n";
			std::copy(path.begin(), path.end(), std::ostream_iterator<int>(file, " "));
			file << "\n";
		}
		else
		{
			file << "Nie znaleziono sciezki. -- Algorytm CUDA BFS Atomowe operacje z shared memory\n";
		}
		delete[] h_prev;
	}
	else
	{
		file << "Nie jest mozliwe znalezienie sciezki tym sposobem. -- Algorytm CUDA BFS Atomowe operacje z shared memory\n";
	}
	return cudaStatus;
}
hipError_t BFSAtomicOppGlobalVersion(thrust::device_vector<int> d_startEdge, thrust::device_vector<int> d_endEdge, thrust::device_vector<int> d_queue, thrust::device_vector<int> d_size_of_old_queue, bool* d_change, int size, int block_size, int num_blocks, thrust::device_vector<int> begin_list, thrust::device_vector<int> distance, int number_of_edges, int iteration, int v, thrust::device_vector<int> d_prev, ofstream& file, int u)
{
	hipError_t cudaStatus = hipSuccess;
	bool change = true;
	int zero = 0, distance_v = MAX_DISTANCE;
	int* pos;
	thrust::device_vector<int> d_new_queue(size);
	thrust::fill(d_new_queue.begin(), d_new_queue.begin() + size, -1);
	cudaStatus = hipMalloc((void**)&pos, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for pos: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}
	// petla ktora dziala do momentu az ostatni wierzcholek jest nieodwiedzony i jest jeszcze nieodwiedzony sasiad
	while (change && distance_v == MAX_DISTANCE) {
		change = false;
		hipMemcpy(d_change, &change, sizeof(bool), hipMemcpyHostToDevice);
		hipMemcpy(pos, &zero, sizeof(bool), hipMemcpyHostToDevice);

		// wlaczam BFS z atomowymi operacjami i globalna kolejka
		BFSAtomicOppGlobalMemory << <num_blocks, block_size >> > (
			thrust::raw_pointer_cast(d_queue.data()),
			thrust::raw_pointer_cast(d_new_queue.data()),
			pos,
			size,
			thrust::raw_pointer_cast(d_startEdge.data()),
			thrust::raw_pointer_cast(d_endEdge.data()),
			thrust::raw_pointer_cast(begin_list.data()),
			thrust::raw_pointer_cast(distance.data()),
			d_change,
			d_startEdge.size(),
			thrust::raw_pointer_cast(d_size_of_old_queue.data()),
			iteration++,
			thrust::raw_pointer_cast(d_prev.data())
			);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
			return cudaStatus;

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)
			return cudaStatus;

		hipMemcpy(&change, d_change, sizeof(bool), hipMemcpyDeviceToHost);
		hipMemcpy(d_size_of_old_queue.data().get(), pos, sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(&distance_v, thrust::raw_pointer_cast(&(distance[v])), sizeof(int), hipMemcpyDeviceToHost);
		thrust::copy(d_new_queue.begin(), d_new_queue.end(), d_queue.begin());
		thrust::fill(d_new_queue.begin(), d_new_queue.begin() + size, -1);
	}
	vector<int> path;
	int* h_prev = new int[size + 1];
	hipMemcpy(h_prev, thrust::raw_pointer_cast(d_prev.data()), (size + 1) * sizeof(int), hipMemcpyDeviceToHost);

	// Znajdz i wypisz sciezke
	findPath(h_prev, u, v, path);
	if (!path.empty())
	{
		file << "\n Global Atomic Opperations BFS\n";
		std::copy(path.begin(), path.end(), std::ostream_iterator<int>(file, " "));
		file << "\n";
	}
	else
	{
		file << "Nie znaleziono sciezki. -- Algorytm CUDA BFS Atomowe operacje globalna pamiec\n";
	}
	delete[] h_prev;
	return cudaStatus;
}